#include "hip/hip_runtime.h"
#include "convert.cuh"

namespace cuda_common
{

	__device__ void YUV2RGB2(uint32* yuvi, float* red, float* green, float* blue)
	{
		float luma, chromaCb, chromaCr;

		// Prepare for hue adjustment
		luma = (float)yuvi[0];
		chromaCb = (float)((int32)yuvi[1] - 512.0f);
		chromaCr = (float)((int32)yuvi[2] - 512.0f);


		// Convert YUV To RGB with hue adjustment
		*red = MUL(luma, constHueColorSpaceMat2[0]) +
			MUL(chromaCb, constHueColorSpaceMat2[1]) +
			MUL(chromaCr, constHueColorSpaceMat2[2]);
		*green = MUL(luma, constHueColorSpaceMat2[3]) +
			MUL(chromaCb, constHueColorSpaceMat2[4]) +
			MUL(chromaCr, constHueColorSpaceMat2[5]);
		*blue = MUL(luma, constHueColorSpaceMat2[6]) +
			MUL(chromaCb, constHueColorSpaceMat2[7]) +
			MUL(chromaCr, constHueColorSpaceMat2[8]);

	}

	__device__ unsigned char clip_v(int x, int min_val, int  max_val)
	{
		if (x > max_val)
		{
			return max_val;
		}
		else if (x < min_val)
		{
			return min_val;
		}
		else
		{
			return x;
		}
	}

	// CUDA kernel for outputing the final RGB output from NV12;

	extern "C"
		__global__ void CUDAToBGR_drvapi(uint32 * dataY, uint32 * dataUV, size_t pitchY, size_t pitchUV, unsigned char* dstImage, int width, int height)
	{

		int32 x, y;

		// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
		x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
		y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= width)
		{
			return;
		}

		if (y >= height)
		{
			return;
		}

		uint32 yuv101010Pel[2];
		uint8* srcImageU8_Y = (uint8*)dataY;
		uint8* srcImageU8_UV = (uint8*)dataUV;

		// Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
		// if we move to texture we could read 4 luminance values
		yuv101010Pel[0] = (srcImageU8_Y[y * pitchY + x]) << 2;
		yuv101010Pel[1] = (srcImageU8_Y[y * pitchY + x + 1]) << 2;

		int32 y_chroma = y >> 1;

		if (y & 1)  // odd scanline ?
		{
			uint32 chromaCb;
			uint32 chromaCr;

			chromaCb = srcImageU8_UV[y_chroma * pitchUV + x];
			chromaCr = srcImageU8_UV[y_chroma * pitchUV + x + 1];

			if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
			{
				chromaCb = (chromaCb + srcImageU8_UV[(y_chroma + 1) * pitchUV + x] + 1) >> 1;
				chromaCr = (chromaCr + srcImageU8_UV[(y_chroma + 1) * pitchUV + x + 1] + 1) >> 1;
			}

			yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
			yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

			yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
			yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
		}
		else
		{
			yuv101010Pel[0] |= ((uint32)srcImageU8_UV[y_chroma * pitchUV + x] << (COLOR_COMPONENT_BIT_SIZE + 2));
			yuv101010Pel[0] |= ((uint32)srcImageU8_UV[y_chroma * pitchUV + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

			yuv101010Pel[1] |= ((uint32)srcImageU8_UV[y_chroma * pitchUV + x] << (COLOR_COMPONENT_BIT_SIZE + 2));
			yuv101010Pel[1] |= ((uint32)srcImageU8_UV[y_chroma * pitchUV + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
		}

		// this steps performs the color conversion
		uint32 yuvi[6];
		float red[2], green[2], blue[2];

		yuvi[0] = (yuv101010Pel[0] & COLOR_COMPONENT_MASK);
		yuvi[1] = ((yuv101010Pel[0] >> COLOR_COMPONENT_BIT_SIZE) & COLOR_COMPONENT_MASK);
		yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

		yuvi[3] = (yuv101010Pel[1] & COLOR_COMPONENT_MASK);
		yuvi[4] = ((yuv101010Pel[1] >> COLOR_COMPONENT_BIT_SIZE) & COLOR_COMPONENT_MASK);
		yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

		// YUV to RGB Transformation conversion
		YUV2RGB2(&yuvi[0], &red[0], &green[0], &blue[0]);
		YUV2RGB2(&yuvi[3], &red[1], &green[1], &blue[1]);


		dstImage[y * width * 3 + x * 3] = clip_v(blue[0] * 0.25, 0, 255);
		dstImage[y * width * 3 + x * 3 + 3] = clip_v(blue[1] * 0.25, 0, 255);

		dstImage[width * y * 3 + x * 3 + 1] = clip_v(green[0] * 0.25, 0, 255);
		dstImage[width * y * 3 + x * 3 + 4] = clip_v(green[1] * 0.25, 0, 255);

		dstImage[width * y * 3 + x * 3 + 2] = clip_v(red[0] * 0.25, 0, 255);
		dstImage[width * y * 3 + x * 3 + 5] = clip_v(red[1] * 0.25, 0, 255);
	}

	hipError_t setColorSpace2(float hue)
	{

		float hueSin = sin(hue);
		float hueCos = cos(hue);

		float hueCSC[9];
		//CCIR 709
		hueCSC[0] = 1.0f;
		hueCSC[1] = hueSin * 1.57480f;
		hueCSC[2] = hueCos * 1.57480f;
		hueCSC[3] = 1.0;
		hueCSC[4] = (hueCos * -0.18732f) - (hueSin * 0.46812f);
		hueCSC[5] = (hueSin * 0.18732f) - (hueCos * 0.46812f);
		hueCSC[6] = 1.0f;
		hueCSC[7] = hueCos * 1.85560f;
		hueCSC[8] = hueSin * -1.85560f;

		hipError_t cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(constHueColorSpaceMat2), hueCSC, 9 * sizeof(float), 0, hipMemcpyHostToDevice);
		float tmpf[9];
		memset(tmpf, 0, 9 * sizeof(float));
		hipMemcpyFromSymbol(tmpf, HIP_SYMBOL(constHueColorSpaceMat2), 9 * sizeof(float), 0, ::hipMemcpyDefault);
		hipDeviceSynchronize();

		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpyToSymbol failed: %s\n", hipGetErrorString(cudaStatus));
		}

		return cudaStatus;
	}

	hipError_t CUDAToBGR(uint32* dataY, uint32* dataUV, size_t pitchY, size_t pitchUV, unsigned char* d_dstRGB, int width, int height)
	{
		dim3 block(32, 16, 1);
		dim3 grid((width + (2 * block.x - 1)) / (2 * block.x), (height + (block.y - 1)) / block.y, 1);
		CUDAToBGR_drvapi << < grid, block >> > (dataY, dataUV, pitchY, pitchUV, d_dstRGB, width, height);
		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "NV12ToRGB_drvapi launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching NV12ToRGB_drvapi !\n", cudaStatus);
			return cudaStatus;
		}

		return cudaStatus;
	}
}